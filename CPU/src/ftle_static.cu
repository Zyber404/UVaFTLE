#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <time.h>
#include <assert.h>
#include "omp.h"

#include <hip/hip_runtime.h>

#include "ftle.h"
#include "arithmetic.h"
#include "preprocess.h"

#define blockSize 512

int main(int argc, char *argv[]) {

	// Check usage
	if (argc != 7)
	{
		printf("USAGE: ./executable <nDim> <coords_file> <faces_file> <flowmap_file> <t_eval>\n");
		printf("\texecutable:    compute_ftle\n");
		printf("\tnDim:    dimensions of the space (2D/3D)\n");
		printf("\tcoords_file:   file where mesh coordinates are stored.\n");
		printf("\tfaces_file:    file where mesh faces are stored.\n");
		printf("\tflowmap_file:  file where flowmap values are stored.\n");
		printf("\tt_eval:        time when compute ftle is desired.\n");
		printf("\tnth:           number of OpenMP threads to use.\n");
		return 1;
	}

    struct timeval start;
    struct timeval end;
    double time;
    double t_eval = atof(argv[5]);

    int nDim, nVertsPerFace, nPoints, nFaces;

	double *coords;
	double *flowmap;
	int    *faces, *d2_faces;
	
	int    *nFacesPerPoint, *d2_nFacesPerPoint;
	int    *facesPerPoint, *d2_facesPerPoint;

	double *w;
	double *logSqrt;

    double *ftl_matrix;

	/* Initialize mesh original information */
	nDim = atoi(argv[1]);
	if ( nDim == 2 ) 
        nVertsPerFace = 3;    // 2D: faces are triangles
	else {
		if ( nDim == 3) 
            nVertsPerFace = 4; // 3D: faces (volumes) are tetrahedrons
		else 
        { 
            printf("Wrong dimension provided (2 or 3 supported)\n"); 
            return 1;
        }
	}

	/* Read coordinates, faces and flowmap from Python-generated files and generate corresponding GPU vectors */
	int  check_EOF;
        char buffer[255];
        FILE *file;

        // Open file
        file = fopen( argv[2], "r" );
        
	// Get nPoints
        check_EOF = fscanf(file, "%s", buffer);
        if ( check_EOF == EOF )
        {
        	fprintf( stderr, "Error: Unexpected EOF in read_coordinates\n" );
        	exit(-1);
        }
        nPoints = atoi(buffer);
	fclose(file);

	// Get nFaces
	file = fopen( argv[3], "r" );
	check_EOF = fscanf(file, "%s", buffer);
	if ( check_EOF == EOF )
	{
		fprintf( stderr, "Error: Unexpected EOF in read_faces\n" );
		exit(-1);
	}
	nFaces = atoi(buffer);

	// Read coordinates, faces and flowmap values
	printf("Reading mesh points coordinates...                     ");
	coords = (double *) malloc ( sizeof(double) * nPoints * nDim );
	read_coordinates(argv[2], nDim, nPoints, coords); 
	printf("DONE\n");
   	
	printf("Reading mesh faces vertices...                         "); 
	faces = (int *) malloc ( sizeof(int) * nFaces * nVertsPerFace );
	read_faces(argv[3], nDim, nVertsPerFace, nFaces, faces); 
	printf("DONE\n");
	
	printf("Reading mesh flowmap (x, y[, z])...                    ");
	flowmap = (double*) malloc( sizeof(double) * nPoints * nDim ); 
	read_flowmap ( argv[4], nDim, nPoints, flowmap );
	printf("DONE\n"); 

    /* Allocate additional memory at the CPU */
    ftl_matrix     = (double*) malloc( sizeof(double) * nPoints * nDim * nDim  );   
    logSqrt        = (double*) malloc( sizeof(double) * nPoints);   
    w              = (double*) malloc( sizeof(double) * nPoints * nDim );
    nFacesPerPoint = (int *) malloc( sizeof(int) * nPoints ); /* REMARK: nFacesPerPoint accumulates previous nFacesPerPoint */

dim3 block(blockSize);
    int numBlocks = (int) (ceil(    (double)nPoints/(double)block.x)  +1);
    numBlocks = numBlocks/omp_get_num_threads() + 1; 
	dim3 grid_numCoords(numBlocks);
        
	/* Assign faces to vertices and generate nFacesPerPoint and facesPerPoint GPU vectors */
	printf("Setting up relationships between faces and vertices... "); fflush(stdout);
	create_nFacesPerPoint_vector ( nDim, nPoints, nFaces, nVertsPerFace, faces, nFacesPerPoint );
    	facesPerPoint = (int *) malloc( sizeof(int) * nFacesPerPoint[ nPoints - 1 ] );
	hipMalloc( &d2_facesPerPoint, sizeof(int)    *   nFacesPerPoint[ nPoints - 1 ]); 
        hipMalloc( &d2_faces,   sizeof(int)    * nFaces  * nVertsPerFace ); 
        hipMalloc( &d2_nFacesPerPoint, sizeof(int)    * nPoints); 
        hipMemcpy( d2_faces,   faces,   sizeof(int) * nFaces * nVertsPerFace, hipMemcpyHostToDevice ); 
        hipMemcpy( d2_nFacesPerPoint, nFacesPerPoint, sizeof(int) * nPoints, hipMemcpyHostToDevice );
	create_facesPerPoint_vector_GPU<<<(ceil(    (double)nPoints/(double)blockSize)  +1),blockSize>>> ( nDim, nPoints, nFaces, nVertsPerFace, 
                                              d2_faces, d2_nFacesPerPoint, d2_facesPerPoint );
	hipMemcpy( facesPerPoint,   d2_facesPerPoint,   sizeof(int)    * nFacesPerPoint[ nPoints - 1 ], hipMemcpyDeviceToHost );
	printf("DONE\n"); fflush(stdout);

    	/* Solve FTLE */
	double max;
	int nth = atoi(argv[6]);
	printf("Computing FTLE...                                       ");
    	gettimeofday(&start, NULL);

    #pragma omp parallel for default(none) shared(nDim, nPoints, nFaces, nVertsPerFace, coords, flowmap, faces, nFacesPerPoint, facesPerPoint, ftl_matrix, logSqrt, t_eval) private(max) num_threads(nth) schedule(static)
    for ( int ip = 0; ip < nPoints; ip++ )
    {
    	/* Compute gradient, tensors and ATxA based on neighbors flowmap values, then get the max eigenvalue */
	if ( nDim == 2 )
        	compute_gradient_2D ( ip, nVertsPerFace, 
                                      coords, flowmap, faces, nFacesPerPoint, facesPerPoint, 
                                      &ftl_matrix[ip * nDim * nDim]);
	else
        	compute_gradient_3D  ( ip, nVertsPerFace, 
                                       coords, flowmap, faces, nFacesPerPoint, facesPerPoint, 
                                       &ftl_matrix[ip * nDim * nDim]);
	if ( nDim == 2)
	{
		max = max_eigen_2D( ftl_matrix[ip * nDim * nDim], 
				ftl_matrix[ip * nDim * nDim+1], 
				ftl_matrix[ip * nDim * nDim+2], 
				ftl_matrix[ip * nDim * nDim+3]);
	}
	else // 3D
	{
                max = max_eigen_3D( ftl_matrix[ip * nDim * nDim],
                                ftl_matrix[ip * nDim * nDim+1], 
                                ftl_matrix[ip * nDim * nDim+2], 
                                ftl_matrix[ip * nDim * nDim+3], 
				ftl_matrix[ip * nDim * nDim+4],
                                ftl_matrix[ip * nDim * nDim+5],
                                ftl_matrix[ip * nDim * nDim+6],
				ftl_matrix[ip * nDim * nDim+7],
                                ftl_matrix[ip * nDim * nDim+8]);
	}

    	/* ftle <- log(srqt(max(eigen))) */
    	logSqrt[ip] = log(sqrt(max))/t_eval;
   }
   /* Time */
   gettimeofday(&end, NULL);
   printf("DONE\n");

    /* Uncomment to show numerical results */
/*
    for ( int ii = 0; ii < nPoints; ii++ )
    {
        printf("%f\n", logSqrt[ii]);
    }
*/

    /* Show time */   
    time = (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec)/1000000.0;
    printf("Execution time: %f\n", time); fflush(stdout);

    /* Free memory */
    free(coords);
    free(flowmap);
    free(faces);
    free(nFacesPerPoint);
    free(facesPerPoint);
    free(w);
    free(logSqrt);
    free(ftl_matrix);

    return 0;
}
